#include <hip/hip_runtime.h>


__global__ void compute_action_probabilities(float* inputs, float* outputs, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        // Perform computations
        outputs[idx] = tanh(inputs[idx]);
    }
}
